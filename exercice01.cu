
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <stdio.h>

// 1.2.4 Modification des parametres
typedef float ft;
const int sub_parts = 64;
const size_t ds = 1024*1024*sub_parts;
const int count = 22;
const int num_streams = 8;


const float sqrt_2PIf = 2.5066282747946493232942230134974f;
const double sqrt_2PI = 2.5066282747946493232942230134974;
__device__ float gpdf(float val, float sigma) {
  return expf(-0.5f * val * val) / (sigma * sqrt_2PIf);
}

__device__ double gpdf(double val, double sigma) {
  return exp(-0.5 * val * val) / (sigma * sqrt_2PI);
}

//  calcul la moyenne de la densite de probabilite sur un interval de valeurs autour de chaque point.
__global__ void gaussian_pdf(const ft * __restrict__ x, ft * __restrict__ y, const ft mean, const ft sigma, const int n) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < n) {
    ft in = x[idx] - (count / 2) * 0.01f;
    ft out = 0;
    for (int i = 0; i < count; i++) {
      ft temp = (in - mean) / sigma;
      out += gpdf(temp, sigma);
      in += 0.01f;
    }
    y[idx] = out / count;
  }
}

// Verification d'erreur CUDA
#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)

// Calcul du temps sur l'host
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start) {
  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

int main() {
  ft *h_x, *d_x, *h_y, *h_y1, *d_y;
  h_x = (ft *)malloc(ds*sizeof(ft));
  h_y = (ft *)malloc(ds*sizeof(ft));
  h_y1 = (ft *)malloc(ds*sizeof(ft));

  hipMalloc(&d_x, ds*sizeof(ft));
  hipMalloc(&d_y, ds*sizeof(ft));
  cudaCheckErrors("allocation error");

  gaussian_pdf<<<(ds + 255) / 256, 256>>>(d_x, d_y, 0.0, 1.0, ds); // warm-up

  for (size_t i = 0; i < ds; i++) {
    h_x[i] = rand() / (ft)RAND_MAX;
  }
  hipDeviceSynchronize();

  unsigned long long et1 = dtime_usec(0);

  hipMemcpy(d_x, h_x, ds * sizeof(ft), hipMemcpyHostToDevice);
  gaussian_pdf<<<(ds + 255) / 256, 256>>>(d_x, d_y, 0.0, 1.0, ds);
  hipMemcpy(h_y1, d_y, ds * sizeof(ft), hipMemcpyDeviceToHost);
  cudaCheckErrors("non-streams execution error");

  et1 = dtime_usec(et1);
  std::cout << "non-stream elapsed time: " << et1/(float)USECPSEC << std::endl;

#ifdef USE_STREAMS
  cudaMemset(d_y, 0, ds * sizeof(ft));

  unsigned long long et = dtime_usec(0);

  // 1.2.1 Creation des streams
 

  // 1.2.2 Execution des streams
  


  et = dtime_usec(et);

  for (int i = 0; i < ds; i++) {
    if (h_y[i] != h_y1[i]) {
      std::cout << "mismatch at " << i << " was: " << h_y[i] << " should be: " << h_y1[i] << std::endl;
      return -1;
    }
  }

  // 1.2.1 Destruction des streams

  std::cout << "streams elapsed time: " << et/(float)USECPSEC << std::endl;
#endif

  return 0;
}
